#include "hip/hip_runtime.h"
﻿#include ""
#include "hip/hip_runtime.h"

#include <algorithm>
#include <stdio.h>
#include <cstdlib>

#define export _declspec(dllexport)

struct sec {
    unsigned __int64 start = 0;
    int steps = 0;
};

hipError_t findBetweenCuda(const int* under, const int* thousands, unsigned __int64 start, unsigned __int64 end, sec* s);

__global__ void findKernel(const int* under, const int* thousands, int* steps, unsigned __int64 start, unsigned __int64 size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        steps[i] = 1;
        __int64 number = start + i;
        while (number != 4) {
            if (number < 1000)
            {
                number = under[number];
                steps[i]++;
            }
            else
            {
                int separated[10];
                int j = 0;
                while (number >= 1000)
                {
                    separated[j++] = (int)(number % 1000);
                    number /= 1000;
                }
                separated[j] = (int)(number % 1000);
                int nonZero = 0;
                while (separated[nonZero] == 0) ++nonZero;
                number = 0;
                int l = j;
                if (separated[l] != 1)
                    number += under[separated[l]] + 1;
                if (nonZero == j) {
                    number += thousands[--l];
                    continue;
                }
                else
                    number += thousands[--l] - 1;
                while (l > nonZero)
                {
                    if (separated[l] == 0)
                    {
                        --l;
                        continue;
                    }
                    number += 2;
                    if (separated[l] != 1)
                        number += under[separated[l]] + 1;
                    number += thousands[--l] - 1;
                }
                number += 2;
                if (nonZero == 0)
                    number += under[separated[nonZero]];
                else
                {
                    if (separated[nonZero] != 1)
                        number += under[separated[nonZero]] + 1;
                    number += thousands[nonZero - 1];
                }
                steps[i]++;
            }
        }
    }
}

int _blocks;
const int underSize = 1000, thousandsSize = 11;
int _under[underSize], _thousands[thousandsSize];

extern "C" {
    export void prepare(int under[underSize], int thousands[thousandsSize], unsigned int blocks) {
        //printf("preparing\r\n");
        for (int i = 0; i < underSize; ++i)
            _under[i] = under[i];
        for (int i = 0; i < thousandsSize; ++i)
            _thousands[i] = thousands[i];
        _blocks = blocks;
    }

    export sec findBetween(unsigned __int64 start, unsigned __int64 end) {
        sec s;
        hipError_t cudaStatus = findBetweenCuda(_under, _thousands, start, end, &s);
        if (cudaStatus != hipSuccess)
            fprintf(stderr, "findBetweenCuda failed!\r\n");
        return s;
    }

    export void reset() {
        // hipDeviceReset must be called before exiting in order for profilingand
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!\r\n");
        }
    }
}

// Helper function for using CUDA.
hipError_t findBetweenCuda(const int* under, const int* thousands, unsigned __int64 start, unsigned __int64 end, sec* s)
{
    //printf("starting\r\n");
    const unsigned __int64 maxSize = _blocks * 1000;
    unsigned __int64 size = std::min(end - start, maxSize);
    int* steps = (int*)malloc(size * sizeof(int));

    int* dev_under = 0;
    int* dev_thousands = 0;
    int* dev_steps = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\r\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_under, underSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (1)\r\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_thousands, thousandsSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (2)\r\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_under, under, underSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (3)\r\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_thousands, thousands, thousandsSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (4)\r\n");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_steps, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (5)\r\n");
        goto Error;
    }

    while (start < end) {
        //printf("searching\r\n");
        unsigned __int64 b = size == maxSize ? _blocks : size / 1000 + 1;
        // Launch a kernel on the GPU with one thread for each element.
        // Ignore the E0029 "expected an expression" error
        findKernel << <b, 1000 >> > (dev_under, dev_thousands, dev_steps, start, size);
        //printf("finishing\r\n");

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "findKernel launch failed: %s\r\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching findKernel!\r\n", cudaStatus);
            goto Error;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(steps, dev_steps, size * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed! (6)\r\n");
            goto Error;
        }

        /*for (int i = 0; i < size; ++i)
            printf("%d: %d\r\n", i, steps[i]);
        printf("\r\n\r\n");*/

        for (int i = 0; i < size; ++i)
            if (steps[i] > s->steps) {
                s->steps = steps[i];
                s->start = start + i;
            }

        start += size;
        size = std::min(end - start, maxSize);
    }

Error:
    hipFree(dev_under);
    hipFree(dev_thousands);
    hipFree(dev_steps);
    free(steps);

    /*fprintf(stderr, hipGetErrorString(cudaStatus));
    printf("\r\n");*/

    return cudaStatus;
}